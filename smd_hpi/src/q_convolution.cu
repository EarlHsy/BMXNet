/*!
 * Copyright (c) 2016 by Contributors
 * \file q_convolution.cu
 * \brief Quantized CONV operator
 * \author HPI-DeepLearning
*/

#include "./q_convolution-inl.h"
#include <mshadow/tensor.h>

namespace mshadow {
// !deprecated! will be removed later
// namespace cuda {
//#include "./xnor_kernels.h"
// inline void QConvolutionForward(const Tensor<gpu, 2, float> &wmat,
//                                 const Tensor<gpu, 2, float> &in_col,
//                                 const Tensor<gpu, 2, float> &temp_dst) {
// 	//======== TODO: able to support arbitrary input channel size ==========//
// 	CHECK_EQ(in_col.size(0) % BITS_PER_BINARY_WORD, 0) << "input channel number for binary convolution layer is not divisible by 32.";
                            
// 	//get matrix dimension		
// 	int m, n, k;
// 	int basic_factor_nchannel_input = BITS_PER_BINARY_WORD;
// 	m = wmat.size(0);
// 	n = wmat.size(1);
// 	k = in_col.size(1);	
	
// 	//check matrix dims:
// 	// 	wmat.size(1) should equal in_col.size(0)
// 	//	temp_dst should have dims (m x k)
// 	CHECK_EQ((int)wmat.size(1), (int)in_col.size(0));
// 	CHECK_EQ((int)temp_dst.size(0), (int)wmat.size(0));
// 	CHECK_EQ((int)temp_dst.size(1), (int)in_col.size(1));
	
// 	hipStream_t stream = Stream<gpu>::GetStream(temp_dst.stream_);
	
// 	//set memory
// 	float *fA = wmat.dptr_; 
// 	float *fB = in_col.dptr_;
// 	float *fC = temp_dst.dptr_;	
			
// 	//set bit memory
// 	//!!NOTE!! here we save 32 float numbers into one binary word
// 	BINARY_WORD *Aconc, *Bconc;
// 	hipMalloc(&Aconc, m*n/basic_factor_nchannel_input*sizeof(int));
// 	hipMalloc(&Bconc, n*k/basic_factor_nchannel_input*sizeof(int));				
	
// 	//concatinates matrix (m x n) -> (m x n/32)
// 	// kMaxThreadsPerBlock defined in "mxnet/mshadow/mshadow/cuda/tensor_gpu-inl.cuh"
// 	int threads_per_block = kMaxThreadsPerBlock;
// 	int blocks_per_grid = m * n / (threads_per_block * basic_factor_nchannel_input) + 1;
// 	concatenate_rows_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(fA, Aconc, m * n / basic_factor_nchannel_input);

// 	//concatinates matrix (n x k) -> (n/32 x k)
// 	threads_per_block = kMaxThreadsPerBlock;
// 	blocks_per_grid = k / threads_per_block + 1;
// 	concatenate_cols_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(fB, Bconc, n, k);
// 	hipDeviceSynchronize();
	
// 	//perform xnor gemm
// 	threads_per_block = BLOCK_SIZE_XNOR;
// 	dim3 blockDim(threads_per_block, threads_per_block);
// 	dim3 gridDim(k / threads_per_block + 1, m / threads_per_block + 1);
// 	xnor_gemm<<<gridDim, blockDim, 0, stream>>>(Aconc, Bconc, fC, m, n / basic_factor_nchannel_input, k);		
// 	hipDeviceSynchronize();	
			
// 	hipFree(Aconc);
// 	hipFree(Bconc);
// }
// }  // namespace cuda

	inline void QConvolutionForward(int m, int n, int k,
									mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
									Tensor<gpu, 1, float> &workspace,
									const Tensor<gpu, 2, float> &in_col,
									Tensor<gpu, 2, float> &temp_dst) {
		CHECK(false) << "cuda with pre-binarized weights not implemented";
	}

	inline void QConvolutionForward(int m, int n, int k,
									const Tensor<gpu, 2, float> &wmat,
									Tensor<gpu, 1, float> &workspace,
									const Tensor<gpu, 2, float> &in_col,
									Tensor<gpu, 2, float> &temp_dst) {
		//!deprecated! will be removed later
		//cuda::QConvolutionForward(wmat, in_col, temp_dst);
	}

	template<typename DType>
	inline void QConvolutionForward(int m, int n, int k,
									const Tensor<gpu, 2, DType> &wmat,
									Tensor<gpu, 1, DType> &workspace,
									const Tensor<gpu, 2, DType> &in_col,
									Tensor<gpu, 2, DType> &temp_dst) {
		CHECK(false) << "only float supported";
	}

	template<typename DType>
	inline void QConvolutionForward(int m, int n, int k,
									mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
									Tensor<gpu, 1, DType> &workspace,
									const Tensor<gpu, 2, DType> &in_col,
									Tensor<gpu, 2, DType> &temp_dst) {
		CHECK(false) << "only float supported";
	}
} // namespace mshadow

namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(QConvolutionParam param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new QConvolutionOp<gpu, DType>(param);
  })
  return op;
}

}  // namespace op
}  // namespace mxnet

