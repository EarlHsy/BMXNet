/*!
 * Copyright (c) 2016 by Contributors
 * \file q_convolution.cu
 * \brief Quantized CONV operator
 * \author HPI-DeepLearning
*/

#include "./q_convolution-inl.h"
#include <mshadow/tensor.h>

namespace mshadow {

	inline void QConvolutionForward(int m, int n, int k,
									mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
									Tensor<gpu, 1, float> &workspace,
									const Tensor<gpu, 2, float> &in_col,
									Tensor<gpu, 2, float> &temp_dst) {
		CHECK(false) << "cuda with pre-binarized weights not implemented";
	}

	inline void QConvolutionForward(int m, int n, int k,
									const Tensor<gpu, 2, float> &wmat,
									Tensor<gpu, 1, float> &workspace,
									const Tensor<gpu, 2, float> &in_col,
									Tensor<gpu, 2, float> &temp_dst) {
		//!deprecated! will be removed later
		//cuda::QConvolutionForward(wmat, in_col, temp_dst);
	}

	template<typename DType>
	inline void QConvolutionForward(int m, int n, int k,
									const Tensor<gpu, 2, DType> &wmat,
									Tensor<gpu, 1, DType> &workspace,
									const Tensor<gpu, 2, DType> &in_col,
									Tensor<gpu, 2, DType> &temp_dst) {
		CHECK(false) << "only float supported";
	}

	template<typename DType>
	inline void QConvolutionForward(int m, int n, int k,
									mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
									Tensor<gpu, 1, DType> &workspace,
									const Tensor<gpu, 2, DType> &in_col,
									Tensor<gpu, 2, DType> &temp_dst) {
		CHECK(false) << "only float supported";
	}
} // namespace mshadow

namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(QConvolutionParam param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new QConvolutionOp<gpu, DType>(param);
  })
  return op;
}

}  // namespace op
}  // namespace mxnet

