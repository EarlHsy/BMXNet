/*!
 * Copyright (c) 2016 by Contributors
 * \file q_fully_connected.cu
 * \brief Quantized FC operator
 * \author HPI-DeepLearning
*/
#include "./q_fully_connected-inl.h"
#include <mshadow/tensor.h>

namespace mshadow {
// !deprecated! will be removed later
// namespace cuda {
// #include "./xnor_kernels.h"
// inline void QFullyConnectedForward(const Tensor<gpu, 2, float> &data,
//                                 const Tensor<gpu, 2, float> &wmat,
//                                 const Tensor<gpu, 2, float> &out) {
                                	    
// 	//======== TODO: able to support arbitrary input channel size ==========//
// 	CHECK_EQ(data.size(1) % BITS_PER_BINARY_WORD, 0) << "input channel number for binary fully_connected layer is not divisible by 32.";
                            
// 	//get matrix dimension		
// 	int m, n, k;
// 	int basic_factor_nchannel_input = BITS_PER_BINARY_WORD;
// 	m = data.size(0);
// 	n = data.size(1);
// 	k = wmat.size(1);	
	
// 	//check matrix dims:
// 	// 	data.size(1) should equal wmat.size(0)
// 	//	out should have dims (m, k)
// 	CHECK_EQ((int)data.size(1), (int)wmat.size(0));
// 	CHECK_EQ((int)out.size(0), (int)data.size(0));
// 	CHECK_EQ((int)out.size(1), (int)wmat.size(1));
	
// 	hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
	
// 	//set memory
// 	float *fA = data.dptr_; 
// 	float *fB = wmat.dptr_;
// 	float *fC = out.dptr_;	
			
// 	//set bit memory
// 	//!!NOTE!! here we save 32 float numbers into one binary word
// 	BINARY_WORD *Aconc, *Bconc;
// 	hipMalloc(&Aconc, m*n/basic_factor_nchannel_input*sizeof(int));
// 	hipMalloc(&Bconc, n*k/basic_factor_nchannel_input*sizeof(int));				
	
// 	//concatinates matrix (m x n) -> (m x n/32)
// 	// kMaxThreadsPerBlock defined in "mxnet/mshadow/mshadow/cuda/tensor_gpu-inl.cuh"
// 	int threads_per_block = kMaxThreadsPerBlock;
// 	int blocks_per_grid = m * n / (threads_per_block * basic_factor_nchannel_input) + 1;
// 	concatenate_rows_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(fA, Aconc, m * n / basic_factor_nchannel_input);

// 	//concatinates matrix (n x k) -> (n/32 x k)
// 	threads_per_block = kMaxThreadsPerBlock;
// 	blocks_per_grid = k / threads_per_block + 1;
// 	concatenate_cols_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(fB, Bconc, n, k);
// 	hipDeviceSynchronize();
	
// 	//perform xnor gemm
// 	threads_per_block = BLOCK_SIZE_XNOR;
// 	dim3 blockDim(threads_per_block, threads_per_block);
// 	dim3 gridDim(k / threads_per_block + 1, m / threads_per_block + 1);
// 	xnor_gemm<<<gridDim, blockDim, 0, stream>>>(Aconc, Bconc, fC, m, n / basic_factor_nchannel_input, k);		
// 	hipDeviceSynchronize();	
			
// 	hipFree(Aconc);
// 	hipFree(Bconc);
// }
// }  // namespace cuda

  inline void QFullyConnectedForward(int m, int n, int k,
                                     const Tensor<gpu, 2, float> &data,
                                     Tensor<gpu, 1, float> &workspace,
                                     mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
                                     Tensor<gpu, 2, float> &out) {
    CHECK(false) << "cuda with pre-binarized weights not implemented";
  }

  inline void QFullyConnectedForward(int m, int n, int k,
                                     const Tensor<gpu, 2, float> &data,
                                     Tensor<gpu, 1, float> &workspace,
                                     const Tensor<gpu, 2, float> &wmat,
                                     Tensor<gpu, 2, float> &out) {
    // !deprecated! will be removed later
    //cuda::QFullyConnectedForward(data, wmat, out);
  }

  template<typename DType>
  inline void QFullyConnectedForward(int m, int n, int k,
                                     const Tensor<gpu, 2, DType> &data,
                                     Tensor<gpu, 1, DType> &workspace,
                                     mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
                                     Tensor<gpu, 2, DType> &out) {
    CHECK(false) << "only float supported";
  }

  template<typename DType>
  inline void QFullyConnectedForward(int m, int n, int k,
                                     const Tensor<gpu, 2, DType> &data,
                                     Tensor<gpu, 1, DType> &workspace,
                                     const Tensor<gpu, 2, DType> &wmat,
                                     Tensor<gpu, 2, DType> &out) {
    CHECK(false) << "only float supported";
  }
} // namespace mshadow


namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(QFullyConnectedParam param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new QFullyConnectedOp<gpu, DType>(param);
  })
  return op;
}
}  // namespace op
}  // namespace mxnet
