
#include <hip/hip_runtime.h>
// #include "./xnor_kernels.h"

// // standard gemm example
// __global__ void gemm(float* A, float* B, float* C, int m, int n, int k) {

//     // Block row and column
//     int blockRow = blockIdx.y;
//     int blockCol = blockIdx.x;
    
//     // Thread row and column within Csub
//     int row = threadIdx.y;
//     int col = threadIdx.x;

//     // Each thread block computes one sub-matrix Csub of C
//     float* Csub = &C[BLOCK_SIZE_XNOR * k * blockRow + BLOCK_SIZE_XNOR * blockCol];

//     // Shared memory used to store Asub and Bsub respectively
//     __shared__ float As[BLOCK_SIZE_XNOR][BLOCK_SIZE_XNOR];
//     __shared__ float Bs[BLOCK_SIZE_XNOR][BLOCK_SIZE_XNOR];
    
//     // Each thread computes one element of Csub
//     // by accumulating results into Cvalue
//     // BLOCK_SIZE_XNOR = 16 -> 256 threads, one per Csub element
//     float Cvalue = 0.0;
    
//     // Loop over all the sub-matrices of A and B that are
//     // required to compute Csub
//     // Multiply each pair of sub-matrices together
//     // and accumulate the results
//     for (int i = 0; i < (n / BLOCK_SIZE_XNOR); ++i) {
    
//         // Get sub-matrix Asub of A
//         float* Asub = &A[BLOCK_SIZE_XNOR * blockRow * n + BLOCK_SIZE_XNOR * i];
        
//         // Get sub-matrix Bsub of B
//         float* Bsub = &B[BLOCK_SIZE_XNOR * k * i + BLOCK_SIZE_XNOR * blockCol];
        
//         // Load Asub and Bsub from device memory to shared memory
//         // Each thread loads one element of each sub-matrix
//         As[row][col] = Asub[row*n+col];
//         Bs[row][col] = Bsub[row*k+col];
    
//         // Synchronize to make sure the sub-matrices are loaded
//         // before starting the computation
//         __syncthreads();
        
//         // Multiply Asub and Bsub together
//         for (int j = 0; j < BLOCK_SIZE_XNOR; ++j) Cvalue += As[row][j] * Bs[j][col]; 
        
//         // Synchronize to make sure that the preceding
//         // computation is done before loading two new
//         // sub-matrices of A and B in the next iteration
//         __syncthreads();
//     }
    
//     // Write Csub to device memory
//     // Each thread writes one element
//     if(col + blockCol* BLOCK_SIZE_XNOR< k && row + blockRow* BLOCK_SIZE_XNOR< m) Csub[row*k+col] = Cvalue;
// }

// // 32 single float array ->  32 bits BINARY_WORD
// __device__ BINARY_WORD concatenate(float* array)
// {
//     BINARY_WORD rvalue=0;
//     BINARY_WORD sign;

//     for (int i = 0; i < BITS_PER_BINARY_WORD; i++)
//     {
//         sign = (array[i]>=0);
//         rvalue = rvalue | (sign<< (i));
//     }
    
//     return rvalue;
// }

// //concatinate in standard directions: (ROW_top->ROW_down {COL_left->COL_right} )
// __global__ void concatenate_rows_kernel(float *a, BINARY_WORD *b, int size)
// { 
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if(i<size) b[i] = concatenate(&a[i*BITS_PER_BINARY_WORD]);
// }

// //concatinate column, processing directions: (COL_left->COL_right {ROW_top->ROW_down} ) 
// __global__ void concatenate_cols_kernel(float *a, BINARY_WORD *b, int n, int k)
// {   

//     int j = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if(j<k){        
//         for(int i=0; i<n; i+=BITS_PER_BINARY_WORD){
//         	float * array = new float[BITS_PER_BINARY_WORD];
            
//             for(int bit=0; bit<BITS_PER_BINARY_WORD;bit++) 
//             	array[bit] = a[j + k*(i+bit)];
            
//             b[j+k*i/BITS_PER_BINARY_WORD]=concatenate(array); 
//             delete[] array;
//         }         
//     }
// }

// // CUDA tutorial: http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
// // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory
// // A is shape (m,n), B is shape (n,k) and C is shape (m,k)
// __global__ void xnor_gemm(BINARY_WORD* A, BINARY_WORD* B, float* C, int m, int n, int k) {
    
//     // Block row and column
//     int blockRow = blockIdx.y;
//     int blockCol = blockIdx.x;
    
//     // Thread row and column within Csub
//     int row = threadIdx.y;
//     int col = threadIdx.x;

//     // Each thread block computes one sub-matrix Csub of C
//     float* Csub = &C[BLOCK_SIZE_XNOR * k * blockRow + BLOCK_SIZE_XNOR * blockCol];

//     // Shared memory used to store Asub and Bsub respectively
//     __shared__ BINARY_WORD As[BLOCK_SIZE_XNOR][BLOCK_SIZE_XNOR];
//     __shared__ BINARY_WORD Bs[BLOCK_SIZE_XNOR][BLOCK_SIZE_XNOR];
    
//     // Each thread computes one element of Csub
//     // by accumulating results into Cvalue
//     // BLOCK_SIZE_XNOR = 16 -> 256 threads, one per Csub element
//     BINARY_WORD Cvalue = 0;
    
//     // Loop over all the sub-matrices of A and B that are
//     // required to compute Csub
//     // Multiply each pair of sub-matrices together
//     // and accumulate the results
//     for (int i = 0; i < (n / BLOCK_SIZE_XNOR); ++i) {
    
//         // Get sub-matrix Asub of A
//         BINARY_WORD* Asub = &A[BLOCK_SIZE_XNOR * blockRow * n + BLOCK_SIZE_XNOR * i];
        
//         // Get sub-matrix Bsub of B
//         BINARY_WORD* Bsub = &B[BLOCK_SIZE_XNOR * k * i + BLOCK_SIZE_XNOR * blockCol];
        
//         // Load Asub and Bsub from device memory to shared memory
//         // Each thread loads one element of each sub-matrix
//         As[row][col] = Asub[row*n+col];
//         Bs[row][col] = Bsub[row*k+col];
    
//         // Synchronize to make sure the sub-matrices are loaded
//         // before starting the computation
//         __syncthreads();
        
//         // Multiply Asub and Bsub together
//         // apply xnor and popcount: 
//         //CUDA has population count intrinsics for both 32-bit and 64-bit types. (__popc() and __popcll())
//         for (int j = 0; j < BLOCK_SIZE_XNOR; ++j) Cvalue += __popc(~(As[row][j]^Bs[j][col]));
        
//         // Synchronize to make sure that the preceding
//         // computation is done before loading two new
//         // sub-matrices of A and B in the next iteration
//         __syncthreads();
//     }
    
//     // Write Csub to device memory
//     // Each thread writes one element    
//     if(col + blockCol* BLOCK_SIZE_XNOR< k && row + blockRow* BLOCK_SIZE_XNOR< m) Csub[row*k+col] = (float)Cvalue;
// }
